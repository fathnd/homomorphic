
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensorCopy.cu"
#else

THC_API void
THCTensor_(copy)(THCState* state, THCTensor* dst, THCTensor* src) {
  if (dst == src) return;
  THC_copyTensor<real, real, THCTensor, THCTensor>(state, dst, src);
}

THC_API void
THCTensor_(copyIgnoringOverlaps)(THCState* state, THCTensor* dst, THCTensor* src) {
  // Called when we are copying into an overlapping index `dst`, but
  // we don't care which writer wins. Hacky but it works.
  // This is itself invoked by pointwiseApply2 / THCTensor_copy in
  // case that there are write overlaps.
  // FIXME: really, overlapping writes should be illegal/an error in Torch
  THC_pointwiseApply2<real,
                      real>(
    state, dst, src,
    CopyOp<real,
           real>(),
    ReadOnly, /* ignore overwrites */
    ReadOnly);
}

#define IMPLEMENT_THC_CUDA_TENSOR_COPY(TYPEC, TYPECUDA, SCALARC)        \
  THC_API void                                                          \
  THCTensor_(copyCuda##TYPEC)(THCState *state,                          \
                              THCTensor *self,                          \
                              THCuda##TYPECUDA##Tensor *src) {          \
    THC_copyTensor<real, SCALARC, THCTensor, THCuda##TYPECUDA##Tensor>(state, self, src); \
  }

IMPLEMENT_THC_CUDA_TENSOR_COPY(Byte, Byte, uint8_t)
IMPLEMENT_THC_CUDA_TENSOR_COPY(Char, Char, int8_t)
IMPLEMENT_THC_CUDA_TENSOR_COPY(Short, Short, int16_t)
IMPLEMENT_THC_CUDA_TENSOR_COPY(Int, Int, int32_t)
IMPLEMENT_THC_CUDA_TENSOR_COPY(Long, Long, int64_t)
// THCudaTensor aka the non-existent THCudaFloatTensor
IMPLEMENT_THC_CUDA_TENSOR_COPY(Float, , float)
IMPLEMENT_THC_CUDA_TENSOR_COPY(Double, Double, double)
#ifdef CUDA_HALF_TENSOR
IMPLEMENT_THC_CUDA_TENSOR_COPY(Half, Half, half)
#endif

#undef IMPLEMENT_THC_CUDA_TENSOR_COPY

#endif
